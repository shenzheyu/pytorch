#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/Utils.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/CUDAGeneratorImpl.h>
#include <c10/macros/Macros.h>
#include <hiprand/hiprand_kernel.h>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>

#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_masked_scale_native.h>
#include <ATen/ops/empty_like.h>
#include <ATen/ops/native_dropout_backward_native.h>
#include <ATen/ops/ones_like.h>
#include <ATen/ops/zeros_like.h>
#endif

namespace at::native {

namespace {

// philox generates 128 bits of randomness at a time. Kernel uses this
// explicitly by putting suitably transformed result into float4 for all members
// of float4 to be consumed UNROLL has to be 4. Don't change! Note: VEC <= 4
// (and in most real-world cases will be 4), so same logic applies.
const int UNROLL = 4;

template <
    typename scalar_t,
    typename accscalar_t,
    typename IndexType,
    int ADims,
    int VEC,
    typename mask_t>
#if __CUDA_ARCH__ >= 350 || defined(USE_ROCM)
C10_LAUNCH_BOUNDS_2(256, 4)
#endif
__global__ void fused_dropout_kernel_vec(
    at::cuda::detail::TensorInfo<scalar_t, IndexType> a,
    at::cuda::detail::TensorInfo<scalar_t, IndexType> b,
    at::cuda::detail::TensorInfo<scalar_t, IndexType> c,
    at::cuda::detail::TensorInfo<scalar_t, IndexType> d,
    at::cuda::detail::TensorInfo<mask_t, IndexType> e,
    IndexType totalElements,
    accscalar_t p,
    PhiloxCudaState philox_args) {
  // make sure we don't break assumption that we can't have > 4 elements /
  // thread
  static_assert(VEC <= 4, "Value of VEC must be in [2, 4]");

  using LoadT = memory::aligned_vector<scalar_t, VEC>;
  using MaskLoadT = memory::aligned_vector<mask_t, VEC>;

  auto seeds = at::cuda::philox::unpack(philox_args);
  IndexType idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(std::get<0>(seeds), idx, std::get<1>(seeds), &state);

  // Helps align the total number of times hiprand_uniform4 is called by each
  // thread for the same totalElements in the vec=2 and vec=4 cases.
  bool gridxvec_loop_state = 0;
  accscalar_t scale = 1.0 / p;

  float4 rand;

  // Note: Vectorized loads means we'll stride each thread by an additional VEC
  // factor, as we'll load VEC elements at a time
  for (IndexType linearIndex = idx * VEC; linearIndex < totalElements;
       linearIndex += gridDim.x * blockDim.x * VEC) {
    // local storage
    scalar_t input_src[VEC];
    scalar_t bias_src[VEC];
    scalar_t residual_src[VEC];
    // We'll use this to actually cause vectorized loads later
    LoadT* input_value = reinterpret_cast<LoadT*>(&input_src);
    LoadT* bias_value = reinterpret_cast<LoadT*>(&bias_src);
    LoadT* residual_value = reinterpret_cast<LoadT*>(&residual_src);

    // hiprand_uniform_double was pure evil anyway, not doing what it promises,
    // and there's nothing for halfs, so generate float for everything
    //  Note: need a new set of random values per 4 elements -- we'll handle VEC
    //  elements in this thread, so need ceil(VEC / 4) sets of rand.
    if ((VEC == 4) || (gridxvec_loop_state == 0)) {
      rand = hiprand_uniform4(&state);
    } else {
      // sets up the last two values we generated last iteration to be used this
      // iteration.
      rand.x = rand.z;
      rand.y = rand.w;
      gridxvec_loop_state ^= 1;
    }

    rand.x = rand.x < p;
    rand.y = rand.y < p;
    if (VEC == 4) {
      rand.z = rand.z < p;
      rand.w = rand.w < p;
    }

    // Note: We explicitly check for is_contiguous() before launching the
    // vectorized kernel and replace IndexToOffset call with linearIndex to
    // allow vectorization of NHWC (or other) ordering. Single vectorized load
    *input_value = *reinterpret_cast<LoadT*>(&a.data[linearIndex]);
    *bias_value = *reinterpret_cast<LoadT*>(&b.data[linearIndex]);
    *residual_value = *reinterpret_cast<LoadT*>(&c.data[linearIndex]);

    scalar_t r[VEC];
    mask_t mask[VEC];


// Perform the actual computation
#pragma unroll
    for (int ii = 0; ii < VEC; ii++) {
      r[ii] = (input_src[ii] + bias_src[ii]) * (&rand.x)[ii] * scale +
          residual_src[ii];
      // r[ii] = (input_src[ii] + bias_src[ii] * (&rand.x)[ii] * scale;
      mask[ii] = (mask_t)(&rand.x)[ii];
    }
    // Vectorized writes for both mask & result
    *(reinterpret_cast<LoadT*>(&d.data[linearIndex])) =
        *reinterpret_cast<LoadT*>(&r[0]);
    *(reinterpret_cast<MaskLoadT*>(&e.data[linearIndex])) =
        *reinterpret_cast<MaskLoadT*>(&mask[0]);

    // printf(
    //     "idx: %d,\n input_src: %f, %f, %f, %f,\n bias_src: %f, %f, %f, %f,\n residual_src: %f, %f, %f, %f,\n, output: %f, %f, %f, %f,\n mask: %d, %d, %d, %d\n\n",  
    //     idx,
    //     input_src[0],
    //     input_src[1],
    //     input_src[2],
    //     input_src[3],
    //     bias_src[0],
    //     bias_src[1],
    //     bias_src[2],
    //     bias_src[3],
    //     residual_src[0],
    //     residual_src[1],
    //     residual_src[2],
    //     residual_src[3],
    //     r[0],
    //     r[1],
    //     r[2],
    //     r[3],
    //     mask[0],
    //     mask[1],
    //     mask[2],
    //     mask[3]);

    __syncthreads();
  }
}

template <
    typename scalar_t,
    typename accscalar_t,
    typename IndexType,
    int ADims,
    int BDims = ADims,
    typename mask_t>
#if __CUDA_ARCH__ >= 350 || defined(USE_ROCM)
C10_LAUNCH_BOUNDS_2(256, 4)
#endif
__global__ void fused_dropout_kernel(
    at::cuda::detail::TensorInfo<scalar_t, IndexType> a,
    at::cuda::detail::TensorInfo<scalar_t, IndexType> b,
    at::cuda::detail::TensorInfo<scalar_t, IndexType> c,
    at::cuda::detail::TensorInfo<scalar_t, IndexType> d,
    at::cuda::detail::TensorInfo<mask_t, IndexType> e,
    IndexType totalElements,
    accscalar_t p,
    PhiloxCudaState philox_args) {
  auto seeds = at::cuda::philox::unpack(philox_args);
  IndexType idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(std::get<0>(seeds), idx, std::get<1>(seeds), &state);
  accscalar_t scale = 1.0 / p;

  IndexType rounded_size =
      ((totalElements - 1) / (blockDim.x * gridDim.x * UNROLL) + 1) *
      blockDim.x * gridDim.x * UNROLL;
  for (IndexType linearIndex = idx; linearIndex < rounded_size;
       linearIndex += gridDim.x * blockDim.x * UNROLL) {
    // hiprand_uniform_double was pure evil anyway, not doing what it promises,
    // and there's nothing for halfs, so generate float for everything
    float4 rand = hiprand_uniform4(&state);
    scalar_t input_src[UNROLL];
    scalar_t bias_src[UNROLL];
    scalar_t residual_src[UNROLL];
    rand.x = rand.x < p;
    rand.y = rand.y < p;
    rand.z = rand.z < p;
    rand.w = rand.w < p;
    for (int ii = 0; ii < UNROLL; ii++) {
      IndexType li = linearIndex + blockDim.x * gridDim.x * ii;
      if (li < totalElements) {
        // Convert `linearIndex` into an offset of `a`
        const IndexType aOffset =
            cuda::detail::IndexToOffset<scalar_t, IndexType, ADims>::get(
                li, a);
        input_src[ii] = a.data[aOffset];
        bias_src[ii] = b.data[aOffset];
        residual_src[ii] = c.data[aOffset];
      }
    }
    for (int ii = 0; ii < UNROLL; ii++) {
      IndexType li = linearIndex + blockDim.x * gridDim.x * ii;
      if (li < totalElements) {
        // Convert `linearIndex` into an offset of `b`
        const IndexType bOffset =
            cuda::detail::IndexToOffset<scalar_t, IndexType, BDims>::get(li, d);
        d.data[bOffset] =
            (input_src[ii] + bias_src[ii]) * (&rand.x)[ii] * scale +
            residual_src[ii];
        e.data[bOffset] = (mask_t)(&rand.x)[ii];
      }
    }
    __syncthreads();
  }
}

template <typename mask_t, typename scalar_t, typename accscalar_t>
void masked_scale_kernel(
    at::Tensor& ret,
    const at::Tensor& src,
    const at::Tensor& mask,
    accscalar_t scale) {
  auto iter = at::TensorIteratorConfig()
                  .check_all_same_dtype(false)
                  .add_output(ret)
                  .add_input(src)
                  .add_input(mask)
                  .build();

  at::native::gpu_kernel(
      iter,
      [=] GPU_LAMBDA(const scalar_t src_val, const mask_t mask_val)
          -> scalar_t { return (float)mask_val * src_val * scale; });
}

template <typename scalar_t>
int get_vector_size(at::Tensor self, at::Tensor ret, at::Tensor mask) {
  int vec_size = 4;
  // get the vector size
  if (!self.is_non_overlapping_and_dense() ||
      !ret.is_non_overlapping_and_dense() ||
      !mask.is_non_overlapping_and_dense()) {
    vec_size = 1;
  } else {
    vec_size = memory::can_vectorize_up_to<scalar_t>(
        (char*)self.data_ptr());
  }

  // check that we'd have no remainders - prefer a smaller vector size with no
  // remainders over a larger vector and remainder.
  bool can_vectorize = true;
  do {
    can_vectorize = self.numel() % vec_size == 0 &&
        ret.numel() % vec_size == 0 && mask.numel() % vec_size == 0;
    if (!can_vectorize)
      vec_size /= 2;
  } while (vec_size > 1 && !can_vectorize);
  return can_vectorize ? vec_size : 1;
}

template <typename index_type, typename mask_t>
inline void launcher(
    const Tensor& input1,
    const Tensor& bias1,
    const Tensor& residual1,
    const Tensor& input2,
    const Tensor& bias2,
    const Tensor& residual2,
    Tensor& ret1,
    Tensor& mask1,
    Tensor& ret2,
    Tensor& mask2,
    double p,
    const int64_t nelem,
    const PhiloxCudaState rng_engine_inputs,
    dim3 grid,
    dim3 dim_block) {
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  auto stream1 = at::cuda::getStreamFromPool().stream();
  
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      input1.scalar_type(),
      "fused_dropout",
      [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        accscalar_t pa = (accscalar_t)(p);
        auto input1_info =
            cuda::detail::getTensorInfo<scalar_t, index_type>(input1);
        auto bias1_info =
            cuda::detail::getTensorInfo<scalar_t, index_type>(bias1);
        auto residual1_info =
            cuda::detail::getTensorInfo<scalar_t, index_type>(residual1);
        auto ret1_info =
            cuda::detail::getTensorInfo<scalar_t, index_type>(ret1);
        auto mask1_info =
            cuda::detail::getTensorInfo<mask_t, index_type>(mask1);
        input1_info.collapseDims();
        bias1_info.collapseDims();
        residual1_info.collapseDims();
        ret1_info.collapseDims();
        mask1_info.collapseDims(); // ret and mask are collapsed to 1d
                                   // contiguous tensor

        auto input2_info =
            cuda::detail::getTensorInfo<scalar_t, index_type>(input2);
        auto bias2_info =
            cuda::detail::getTensorInfo<scalar_t, index_type>(bias2);
        auto residual2_info =
            cuda::detail::getTensorInfo<scalar_t, index_type>(residual2);
        auto ret2_info =
            cuda::detail::getTensorInfo<scalar_t, index_type>(ret2);
        auto mask2_info =
            cuda::detail::getTensorInfo<mask_t, index_type>(mask2);
        input2_info.collapseDims();
        bias2_info.collapseDims();
        residual2_info.collapseDims();
        ret2_info.collapseDims();
        mask2_info.collapseDims(); // ret and mask are collapsed to 1d
                                   // contiguous tensor

        int vec_size = get_vector_size<scalar_t>(input1, ret1, mask1);

        if (vec_size > 1) {
          switch (vec_size) {
            case 4:
              fused_dropout_kernel_vec<scalar_t, accscalar_t, index_type, 1, 4>
                  <<<grid, dim_block, 0, stream>>>(
                      input1_info,
                      bias1_info,
                      residual1_info,
                      ret1_info,
                      mask1_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              fused_dropout_kernel_vec<scalar_t, accscalar_t, index_type, 1, 4>
                  <<<grid, dim_block, 0, stream1>>>(
                      input2_info,
                      bias2_info,
                      residual2_info,
                      ret2_info,
                      mask2_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              break;
            case 2:
              fused_dropout_kernel_vec<scalar_t, accscalar_t, index_type, 1, 2>
                  <<<grid, dim_block, 0, stream>>>(
                      input1_info,
                      bias1_info,
                      residual1_info,
                      ret1_info,
                      mask1_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              fused_dropout_kernel_vec<scalar_t, accscalar_t, index_type, 1, 2>
                  <<<grid, dim_block, 0, stream1>>>(
                      input2_info,
                      bias2_info,
                      residual2_info,
                      ret2_info,
                      mask2_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              break;
          }
        } else {
          switch (input1_info.dims) {
            case 1:
              fused_dropout_kernel<scalar_t, accscalar_t, index_type, 1>
                  <<<grid, dim_block, 0, stream>>>(
                      input1_info,
                      bias1_info,
                      residual1_info,
                      ret1_info,
                      mask1_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              fused_dropout_kernel<scalar_t, accscalar_t, index_type, 1>
                  <<<grid, dim_block, 0, stream1>>>(
                      input2_info,
                      bias2_info,
                      residual2_info,
                      ret2_info,
                      mask2_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              break;
            default:
              if (!input1.is_contiguous() && !bias1.is_contiguous() &&
                  !residual1.is_contiguous() && ret1.is_contiguous() &&
                  mask1.is_contiguous()) {
                fused_dropout_kernel<scalar_t, accscalar_t, index_type, -1, 1>
                    <<<grid, dim_block, 0, stream>>>(
                        input1_info,
                        bias1_info,
                        residual1_info,
                        ret1_info,
                        mask1_info,
                        nelem,
                        pa,
                        rng_engine_inputs);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              } else {
                fused_dropout_kernel<scalar_t, accscalar_t, index_type, -1>
                    <<<grid, dim_block, 0, stream>>>(
                        input1_info,
                        bias1_info,
                        residual1_info,
                        ret1_info,
                        mask1_info,
                        nelem,
                        pa,
                        rng_engine_inputs);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
              if (!input2.is_contiguous() && !bias2.is_contiguous() &&
                  !residual2.is_contiguous() && ret2.is_contiguous() &&
                  mask2.is_contiguous()) {
                fused_dropout_kernel<scalar_t, accscalar_t, index_type, -1, 1>
                    <<<grid, dim_block, 0, stream1>>>(
                        input2_info,
                        bias2_info,
                        residual2_info,
                        ret2_info,
                        mask2_info,
                        nelem,
                        pa,
                        rng_engine_inputs);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              } else {
                fused_dropout_kernel<scalar_t, accscalar_t, index_type, -1>
                    <<<grid, dim_block, 0, stream1>>>(
                        input2_info,
                        bias2_info,
                        residual2_info,
                        ret2_info,
                        mask2_info,
                        nelem,
                        pa,
                        rng_engine_inputs);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
          }
        }
      });
}

} // anonymous namespace

template <typename mask_t>
std::tuple<Tensor, Tensor, Tensor, Tensor> dropout_cuda(
    CUDAGeneratorImpl* gen,
    const Tensor& input1,
    const Tensor& bias1,
    const Tensor& residual1,
    const Tensor& input2,
    const Tensor& bias2,
    const Tensor& residual2,
    double p) {
  Tensor mask1 = at::empty_like(
      input1, input1.options().dtype(c10::CppTypeToScalarType<mask_t>::value));
  Tensor mask2 = at::empty_like(
      input2, input2.options().dtype(c10::CppTypeToScalarType<mask_t>::value));
  const int64_t nelem = input1.numel();
  // empty tensors should not get here, but just in case, avoid FPE
  // non-training shot-cut
  if (nelem == 0)
    return std::tuple<Tensor, Tensor, Tensor, Tensor>(
        input1.clone(), mask1, input2.clone(), mask2);

  Tensor ret1 = at::empty_like(input1);
  Tensor ret2 = at::empty_like(input2);
  const int64_t block_size = 256;
  unsigned int blocks_per_sm =
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor /
      block_size;
  dim3 dim_block(block_size);
  dim3 grid((nelem + block_size - 1) / block_size);
  grid.x = std::min(
      (unsigned int)at::cuda::getCurrentDeviceProperties()
              ->multiProcessorCount *
          blocks_per_sm,
      grid.x);
  // number of times random will be generated per thread, to offset philox
  // counter in thc random state
  int64_t counter_offset =
      ((nelem - 1) / (block_size * grid.x * UNROLL) + 1) * UNROLL;
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(counter_offset);
  }
  if (cuda::detail::canUse32BitIndexMath(input1)) {
    launcher<unsigned int, mask_t>(
        input1,
        bias1,
        residual1,
        input2,
        bias2,
        residual2,
        ret1,
        mask1,
        ret2,
        mask2,
        p,
        nelem,
        rng_engine_inputs,
        grid,
        dim_block);
  } else {
    launcher<uint64_t, mask_t>(
        input1,
        bias1,
        residual1,
        input2,
        bias2,
        residual2,
        ret1,
        mask1,
        ret2,
        mask2,
        p,
        nelem,
        rng_engine_inputs,
        grid,
        dim_block);
  }
  // TODO: remove the clone() calls
  // ret2 = ret1.clone();
  // mask2 = mask1.clone();
  return std::tuple<Tensor, Tensor, Tensor, Tensor>(ret1, mask1, ret2, mask2);
}

std::tuple<Tensor, Tensor, Tensor, Tensor> native_add_dropout_add_fuse_cuda(
    const Tensor& input1,
    const Tensor& bias1,
    const Tensor& residual1,
    const Tensor& input2,
    const Tensor& bias2,
    const Tensor& residual2,
    double p,
    c10::optional<bool> train) {
  // short-cut for train == false
  if (train.has_value() && !train.value()) {
    auto ret1 = input1.add(bias1).add(residual1);
    auto ret2 = input2.add(bias2).add(residual2);
    return std::make_tuple(
        ret1,
        at::ones_like(
            input1,
            input1.options().dtype(c10::CppTypeToScalarType<bool>::value)),
        ret2,
        at::ones_like(
            input2,
            input2.options().dtype(c10::CppTypeToScalarType<bool>::value)));
  }
  // short-cut
  if (p == 1) {
    // native_dropout_cuda is in derivatives.yaml, so we don't need to add data
    // dependency from output to input for autograd
    auto ret1 = at::zeros_like(input1);
    auto mask1 = at::zeros_like(
        input1, input1.options().dtype(c10::CppTypeToScalarType<bool>::value));
    auto ret2 = at::zeros_like(input2);
    auto mask2 = at::zeros_like(
        input2, input2.options().dtype(c10::CppTypeToScalarType<bool>::value));
    return std::tuple<Tensor, Tensor, Tensor, Tensor>(ret1, mask1, ret1, mask1);
  }

  auto gen = get_generator_or_default<CUDAGeneratorImpl>(
      c10::nullopt, cuda::detail::getDefaultCUDAGenerator());
  double p1m = 1. - p;
  return dropout_cuda<bool>(
      gen, input1, bias1, residual1, input2, bias2, residual2, p1m);
}

template <typename mask_t>
Tensor dropout_backward_cuda(
    const Tensor& grad,
    const Tensor& mask,
    double scale) {
  Tensor ret = at::empty_like(grad, grad.suggest_memory_format());
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      ret.scalar_type(),
      "masked_scale",
      [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        masked_scale_kernel<mask_t, scalar_t>(
            ret, grad, mask, (accscalar_t)scale);
      });
  return ret;
}

std::tuple<Tensor, Tensor, Tensor, Tensor, Tensor, Tensor>
native_add_dropout_add_fuse_2_cuda(
    const Tensor& grad_output1,
    const Tensor& mask1,
    const Tensor& grad_output2,
    const Tensor& mask2,
    double scale) {
  TORCH_CHECK(
      mask1.scalar_type() == at::ScalarType::Bool,
      "Mask should be Bool Scalar Type",
      mask1.scalar_type());
  Tensor grad_input1 = dropout_backward_cuda<bool>(grad_output1, mask1, scale);
  Tensor grad_bias1 = grad_input1;
  Tensor grad_residual1 = grad_output1;
  Tensor grad_input2 = dropout_backward_cuda<bool>(grad_output2, mask2, scale);
  Tensor grad_bias2 = grad_input2;
  Tensor grad_residual2 = grad_output2;
  return std::tuple<Tensor, Tensor, Tensor, Tensor, Tensor, Tensor>(
      grad_input1,
      grad_bias1,
      grad_residual1,
      grad_input2,
      grad_bias2,
      grad_residual2);
}

} // namespace at::native

